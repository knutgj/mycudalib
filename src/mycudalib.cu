#include "hip/hip_runtime.h"
// should work for manual compile
// nvcc --compiler-options '-fPIC' -m 64 -o libmylib.dylib --shared mycudalib.cu

#include <stdio.h> //for io used in hello
#include <hip/hip_complex.h>

#include "mycudalib.h"

const int threadsPerBlock = 256;
  
/* single float element multiplication of A * B into C */
// declare kernel
__global__ void elementMult_kernel(int N, const float* A, const float* B, float* C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] * B[i];
}
void elementMult(int N, const float* A, const float* B, float* C)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    elementMult_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A, B, C);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* destructive single float element multiplication of A * B into B */
// declare kernel
__global__ void elementMultd_kernel(int N, const float* A, float* B)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        B[i] = A[i] * B[i];
}
void elementMultd(int N, const float* A, float* B)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    elementMultd_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A, B);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* complex single float element multiplication of A * B into C */
// declare kernel
__global__ void elementMultc_kernel(int N, const hipFloatComplex* A, const hipFloatComplex* B, hipFloatComplex* C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
      C[i] = hipCmulf(A[i], B[i]);
}
void elementMultc(int N, const hipFloatComplex* A, const hipFloatComplex* B, hipFloatComplex* C)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    elementMultc_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A, B, C);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* destructive complex single float element multiplication of A * B into B */
// declare kernel
__global__ void elementMultcd_kernel(int N, const hipFloatComplex* A, hipFloatComplex* B)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
      B[i] = hipCmulf(A[i], B[i]);
}
void elementMultcd(int N, const hipFloatComplex* A, hipFloatComplex* B)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    elementMultcd_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A, B);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* copies data from real-valued A to complex-valued B */ 
// declare kernel
__global__ void cpycomplex_kernel(int N, const float* A, hipFloatComplex* B)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
      B[i] = make_hipFloatComplex(A[i], 0.0);
}
void cpycomplex(int N, const float* A, hipFloatComplex* B)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    cpycomplex_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A, B);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* copies the real part of complex-valued A to real-valued B */
// declare kernel
__global__ void cpyreal_kernel(int N, const hipFloatComplex* A, float* B)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
      B[i] = hipCrealf(A[i]);
}
void cpyreal(int N, const hipFloatComplex* A, float* B)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    cpyreal_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A, B);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* sets the ith element of vector A to 0 */
// declare kernel
__global__ void setizero_kernel(float* A, int i)
{
      A[i] = 0.0;
}
void setizero(int N, float* A, int i)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    setizero_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, i);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}

/* sets N elements in A to 1 */
// declare kernel
__global__ void settoone_kernel(int N, float* A)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
      A[i] = 1.0;
}
void settoone(int N, float* A)
{
 // invoke kernel
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    settoone_kernel<<<blocksPerGrid, threadsPerBlock>>>(N, A);
    #ifdef _DEBUG
    hipDeviceSynchronize();
    #endif
}


 /* simple hello world to test ok compile and run */
__global__ void hello_k(char *a, int *b) 
{
        a[threadIdx.x] += b[threadIdx.x];
}
int hello()
{
  const int N = 16;
  const int blocksize = 16;
 
  char a[N] = "Hello \0\0\0\0\0\0";
  int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  
  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);
  
  printf("%s", a);
  
  hipMalloc( (void**)&ad, csize ); 
  hipMalloc( (void**)&bd, isize ); 
  hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
  
  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  hello_k<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
  hipFree( ad );
  
  printf("%s\n", a);
  return EXIT_SUCCESS;
}
